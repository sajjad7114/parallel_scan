#include "hip/hip_runtime.h"

#include "scan2.h"
#include "gpuerrors.h"

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define TILEX 1024


__global__ void kernelFunc(float* ad, float* cd, float* lasts) {
	__shared__ float as[TILEX];
	
	float h = ad[TILEX*bx+tx];
	as[tx] = h;
	__syncthreads();
	
	for (int p = 2; p <= TILEX; p *= 2){
		if((tx+1)%p == 0){
			as[tx] += as[tx - p/2];
		}
		__syncthreads();
	}
	
	if(tx+1 == TILEX){
		as[tx] = 0;
	}
	__syncthreads();
	
	for(int p = TILEX; p >1; p/= 2){
		if((tx+1)%p == 0){
			float r = as[tx];
			float l = as[tx - p/2];
			as[tx - p/2] = r;
			as[tx] = r + l;
		}
		__syncthreads();
	}

	h = as[tx] + h;
	cd[TILEX*bx+tx] = h;
	if(tx+1 == TILEX){
		lasts[bx] = h;
	}
}

__global__ void littleFunc(float* ad, float* cd, float* lasts, int tile) {
	__shared__ float as[128];
	
	float h = ad[tile*bx+tx];
	as[tx] = h;
	__syncthreads();
	
	for (int p = 2; p <= tile; p *= 2){
		if((tx+1)%p == 0){
			as[tx] += as[tx - p/2];
		}
		__syncthreads();
	}
	
	if(tx+1 == tile){
		as[tx] = 0;
	}
	__syncthreads();
	
	for(int p = tile; p >1; p/= 2){
		if((tx+1)%p == 0){
			float r = as[tx];
			float l = as[tx - p/2];
			as[tx - p/2] = r;
			as[tx] = r + l;
		}
		__syncthreads();
	}

	h = as[tx] + h;
	cd[tile*bx+tx] = h;
	if(tx+1 == tile){
		lasts[bx] = h;
	}
}

__global__ void merge(float* ad, float* cd, float num) {
	__shared__ float n;
	if(tx == 0){
		n = num;
		if(bx > 0){
			n += cd[bx-1];
		}
	}
	__syncthreads();
	ad[TILEX*bx+tx] += n;
}

void gpuKernel(float* a, float* c,int n) {
	unsigned long long int rounds = n / (1024*1024*32);
	if (rounds < 1){
		rounds = 1;
	}		
	float* ad;
	float* cd;
	float* lasts;
	float* scaned_lasts;
	float* lasts2;
	float* scaned_lasts2;
	float* nextc;
	float next[1] = {0};
	
	HANDLE_ERROR(hipMalloc((void**)&ad, n/rounds * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&cd, n/rounds * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&lasts, n/rounds/TILEX * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&scaned_lasts, n/rounds/TILEX * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&lasts2, n/rounds/TILEX/TILEX * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&scaned_lasts2, n/rounds/TILEX/TILEX * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&nextc, sizeof(float)));
	
	for (unsigned long long int i = 0; i < rounds; i ++){
		HANDLE_ERROR(hipMemcpy(ad, a+(i*n/rounds), n/rounds * sizeof(float), hipMemcpyHostToDevice));
		dim3 dimBlock(TILEX);
		kernelFunc<<< n/rounds/TILEX, dimBlock >>>(ad, cd, lasts);
		kernelFunc<<< n/rounds/TILEX/TILEX, dimBlock >>>(lasts, scaned_lasts, lasts2);
		littleFunc<<< 1, n/rounds/TILEX/TILEX >>>(lasts2, scaned_lasts2, nextc, n/rounds/TILEX/TILEX);
		
		merge<<< n/rounds/TILEX/TILEX, dimBlock >>>(scaned_lasts, scaned_lasts2, next[0]);
		merge<<< n/rounds/TILEX, dimBlock >>>(cd, scaned_lasts, 0);
		float w = next[0];
		hipMemcpy(next, nextc, sizeof(float), hipMemcpyDeviceToHost);
		next[0] += w;
		HANDLE_ERROR(hipMemcpy(c+i*n/rounds, cd, n/rounds * sizeof(float), hipMemcpyDeviceToHost));
	}
}
